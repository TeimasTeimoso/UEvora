
#include <hip/hip_runtime.h>
#include <vector>
#include <random>
#include <limits>
#include <iostream>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <chrono>

#define ITERATIONS 200

struct Point
{
    float x{0}, y{0};
};

using DataFrame = std::vector<Point>;

__global__ void cudaClear(Point *points, const int bytes)
{
    const int index = threadIdx.x;

    if(index < bytes)
        points[index] = {0,0};

    __syncthreads();

}

struct CUDADataFrame
{
    CUDADataFrame(int size): size(size), bytes(size * sizeof(Point))
    {
        hipMalloc((void **)&points, bytes);
    }

    CUDADataFrame(int size, DataFrame df): size(size), bytes(size * sizeof(Point))
    {
        hipMalloc((void **)&points, bytes);
        hipMemcpy(points, df.data(), bytes, hipMemcpyHostToDevice);
    }

    ~CUDADataFrame()
    {
        hipFree(points);
    }

    void clear()
    {
        cudaClear<<<1, size>>>(points, bytes);
    }

    Point *points;
    int size{0};
    int bytes{0};
};

__device__ float square(float val)
{
    return val * val;
}

__device__ float squared_distance(Point first, Point second)
{
    return square(first.x - second.x) + square(first.y - second.y);
}

__global__ void assign_clusters(const Point *data_set, int set_size, const Point *means, Point *new_sums, int k, int *counts)
{

    extern __shared__ float shared_data[];


    // Index of thread in it's own block
    const int local_index = threadIdx.x;
    const int global_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Set all positions to 0 TEST
    //shared_data[global_index] = 0;

    if(global_index >= set_size) return;

    // Load the means value into shared memory.
    if(local_index < k)
    {
        shared_data[local_index] = means[local_index].x;
        shared_data[local_index + k] = means[local_index].y;
    }

    __syncthreads();

    const Point data_point = data_set[global_index];

    float best_distance = 99999999;
    int best_cluster = -1;

    for(int cluster = 0; cluster < k; cluster++)
    {
        Point p = {shared_data[cluster], shared_data[cluster + k]};

        float distance = squared_distance(data_point, p);

        if(distance < best_distance)
        {
            best_distance = distance;
            best_cluster = cluster;
        }
    }

    __syncthreads();

    atomicAdd(&new_sums[best_cluster].x, data_point.x);
    atomicAdd(&new_sums[best_cluster].y, data_point.y);
    atomicAdd(&counts[best_cluster], 1);
}


__global__ void compute_new_means(Point *means, Point *new_sums, int *counts)
{
    const int cluster = threadIdx.x;
    const int count = max(1, counts[cluster]);
    means[cluster].x = new_sums[cluster].x / count;
    means[cluster].y = new_sums[cluster].y / count;
}

void place_random_centroids(DataFrame &data)
{
    std::mt19937 rng(std::random_device{}());
    std::shuffle(data.begin(), data.end(), rng);
}

int main(int argc, char const *argv[])
{
    if(argc < 3)
    {
        std::cerr << "usage: assign_clusters <data-file> <k> [iterations]" << std::endl;
        std::exit(EXIT_FAILURE);
    }

    const auto k = std::atoi(argv[2]);
    const auto n_of_iterations = (argc == 4) ? std::atoi(argv[3]) : ITERATIONS;

    DataFrame df;

    std::ifstream data_file(argv[1]);
    if(!data_file)
    {
        std::cerr << "Could not open file: " << argv[1] << std::endl;
        std::exit(EXIT_FAILURE);
    }

    std::string line;
    while (std::getline(data_file, line)) {
        std::istringstream line_stream(line);   
        Point point;
        int label;
        line_stream >> point.x >> point.y >> label;
        df.push_back(point);
    }   

    const int n_of_elements = df.size();

    CUDADataFrame device_data(n_of_elements, df);

    place_random_centroids(df);

    CUDADataFrame device_means(k, df);

    const int threads = 1024;
    const int blocks = (n_of_elements + threads - 1) / threads;

    const int fine_shared_memory = 3 * threads * sizeof(float);

    CUDADataFrame device_sums(k);
    int *device_counts;
    hipMalloc(&device_counts, k * sizeof(int));
    hipMemset(device_counts, 0, k * sizeof(int));

    const auto start = std::chrono::high_resolution_clock::now();
    for(int iter = 0; iter < n_of_iterations; iter++)
    {
        assign_clusters<<<blocks, threads, fine_shared_memory>>>(device_data.points, n_of_elements, device_means.points, device_sums.points, k, device_counts);

        hipDeviceSynchronize();

        compute_new_means<<<1, k>>>(device_means.points, device_sums.points, device_counts);

        hipDeviceSynchronize();
    }
  	const auto end = std::chrono::high_resolution_clock::now();
  	const auto duration = std::chrono::duration_cast<std::chrono::duration<float>>(end - start);
  	std::cerr << "Time: " << duration.count() << "s" << std::endl;

    hipFree(device_counts);

    DataFrame result(k);
    hipMemcpy(result.data(), device_means.points, device_means.bytes, hipMemcpyDeviceToHost);

    for(int cluster = 0; cluster < k; cluster++)
    {
        std::cout << result[cluster].x << " " << result[cluster].y << std::endl;
    }


    return 0;
}
